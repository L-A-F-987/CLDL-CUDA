#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>


hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            cudaGetErrorString(result));
    assert(result == cudaSuccess);
  }
#endif
  return result;
}

int main(){

  std::cout<<"This program is written to test the processing time of the cublas library\n\n\n";
  

  for(int i = 100;i<30000;i+=500){

    std::cout<<"Performing dot product of two vectors of length:  "<<i<<"\n";

    double *d_a, *d_b;

    const int ds = i;
    
    hipMalloc(&d_a, sizeof(d_a[0])*ds);
    hipMalloc(&d_b, sizeof(d_b[0])*ds);
    
    double *h = new double[ds];
    for (int i = 0; i < ds; i++) h[i] = 5;
    hipMemcpy(d_a, h, sizeof(d_a[0])*ds, hipMemcpyHostToDevice);
    for (int i = 0; i < ds; i++) h[i] = 2;
    hipMemcpy(d_b, h, sizeof(d_b[0])*ds, hipMemcpyHostToDevice);
    
    hipblasHandle_t hd;
    hipblasStatus_t stat = hipblasCreate(&hd);

    int incy = 1;
    int incx = 1;

    double* result;
    double *res = new double;

    hipMalloc(&result,sizeof(res));
    hipMemcpy(result,res,sizeof(result),hipMemcpyHostToDevice);
    
  //timing the dot product function 
    auto start = std::chrono::high_resolution_clock::now();
    hipblasDdot(hd, ds ,d_a ,incx ,d_b ,incy ,result);
    auto total = std::chrono::high_resolution_clock::now()- start;

  //printing the time taken to perform the dot product
    float total_printable = std::chrono::duration_cast<std::chrono::microseconds>(
        total).count();

    float total_in_ms = total_printable/1000;
    float total_in_seconds = total_in_ms/(1000);

    int num_per_sec = 1/total_in_seconds;

    std::cout<<"Time To Calculate Dot--->"<<total_in_ms<<"ms\n";
    std::cout<<"Number Calculable Per Second--->"<<num_per_sec<<"\n\n";

 

    
    
  }


  //for (int i = 0; i < ds; i++) std::cout << h[i] << std::endl;
}

