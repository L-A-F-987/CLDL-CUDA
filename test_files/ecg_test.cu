#include "hip/hip_runtime.h"

#include <chrono>

#include "cldl/Net.h"

#include <iostream>
#include <stdio.h>
#include <thread>

#define _USE_MATH_DEFINES
#include <math.h>


using namespace std; 

//Creating an error and output variable for use later
    double error = 0;
    double output;


//The nlayers should be an integer of the total number of hidden layers required not including the input layer
    const int nLayers = 10;

//Neuron array should hold the number of neurons for each layer, each array element is a
//single input 
    int Neurons_array[nLayers];


    int *nNeurons = Neurons_array;


//setting up initial inputs
    const int nInputs = 500;

    double Array_of_0s_for_initial_inputs[nInputs];

    double *pointer_to_array_of_0s = Array_of_0s_for_initial_inputs;






    int main(int argc, char* argv[]){

    std::cout<<"Made it to the Start :)\n\n";



//Opening the .dat file and the output file
//in the final program this should be replaced with the mic inputs
    FILE *finput = fopen("ecg50hz.dat","rt");
	FILE *foutput = fopen("ecg_filtered.dat","wt");
    FILE *f_gpu_time = fopen("gpu_time_taken.txt","wt");


//generating a network to be used

//Filling Neurons_array with some arbitray numbers to test network
//Setting the output layer to be of size 1
    Neurons_array[0] = nInputs;
    Neurons_array[1] = 1;
    Neurons_array[2] = 1;
    Neurons_array[3] = 1;
    Neurons_array[4] = 1;
    Neurons_array[5] = 1;
    Neurons_array[6] = 1;
    Neurons_array[7] = 1;
    Neurons_array[8] = 1;
    Neurons_array[nLayers-1] = 1;

//Filling Input array with 0s array 

    for(int i = 0; i<= nInputs;i++){
    Array_of_0s_for_initial_inputs[i] = 0;
    }   


//Varifying that the pointer points to the first element of the array



//Creating the Network 
    Net *net;
    net = new Net::Net(nLayers,nNeurons,nInputs);


//Initialises the network with: weights, biases and activation function
// for Weights; W_Zeroes sets to 0 , W_Ones sets to 1 , W_random sets to a randome value
// for Bias; B_None sets to , B_Random sets to a random value
// for activations functions; Act_Sigmoid, Act_Tanh or Act_None
    net->initNetwork(Neuron::W_ONES, Neuron::B_NONE, Neuron::Act_Sigmoid);



//Setting all intial inputs to 0
    net -> setInputs(pointer_to_array_of_0s);


//Setting Learning Rate
    net -> setLearningRate(0.001);



//Setting up a variable that allows for access to read the final output of the network
    Layer *output_layer = net -> getLayer(nLayers-1);
    Neuron *output_neuron = output_layer ->getNeuron(0);
    int number_of_outputs = output_layer ->getnNeurons();


//Getting variable that allows for access to input layer
    Layer *input_layer = net ->getLayer(0);
    Neuron *input_Neuron_0 = input_layer -> getNeuron(0);
    int number_of_inputs = input_layer ->getnNeurons();

//variale to read the first input layer neuron
    double neuron_one_layer_one;


    std::cout<<"Number of Inputs:"<<number_of_inputs<<"\n";
    std::cout<<"Number of Outputs:"<<number_of_outputs<<"\n";
    std::cout<<"Number of Layers:"<<net->getnLayers()<<"\n";
    std::cout<<"Number of Total Neurons:"<<net->getnNeurons()<<"\n";
    std::cout<<"Neurons Array:";

    for(int i = 0;i<nLayers;i++){
    std::cout<<Neurons_array[i]<<",";
    }
    std::cout<<"\n";




    auto start = std::chrono::high_resolution_clock::now();


    for(int i=0;;i++) 
	{
//timer for gpu 
    auto gpu_timer_1_sample_input = std::chrono::high_resolution_clock::now();

//reading the input signal and generating the ref_noise
	double input_signal;		
	if (fscanf(finput,"%lf\n",&input_signal)<1) break;
	double ref_noise = sin(2*M_PI/20*i);

//Updating the inputs to the network
    for(int i = nInputs-1;i>0;i--){
    Array_of_0s_for_initial_inputs[i] = Array_of_0s_for_initial_inputs[i-1];
    }

    Array_of_0s_for_initial_inputs[0] = input_signal;
        
    net -> setInputs(pointer_to_array_of_0s);



//propegating the sample forwards
    net ->propInputs();


//storing output of the function and calculation error
    output = net->getOutput(0);


    error = ref_noise - output;


//Setting the backward error and updating weights
    net->setBackwardError(error);
    net->propErrorBackward();
    net ->updateWeights();



	fprintf(foutput,"%f \n",output);

    auto gpu_timer_1_sample_time = std::chrono::high_resolution_clock::now() - gpu_timer_1_sample_input;
    fprintf(f_gpu_time,"%i \n",gpu_timer_1_sample_time);
	}

    auto elapsed = std::chrono::high_resolution_clock::now() - start;

    long long microseconds_taken = std::chrono::duration_cast<std::chrono::microseconds>(
        elapsed).count();

    
    std::cout<<"Time Taken:     "<<microseconds_taken<<"µs\n";

    fclose(finput);
	fclose(foutput);
    fclose(f_gpu_time);

//fprintf(stderr,"Written the filtered ECG to 'ecg_filtered.dat'\n");


    std::cout<<"Made it to the End :)\n\n\n";
    
    



}